#include <hip/hip_runtime.h>

__global__ void cube(float * d_out, float * d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f + 21;
}

extern "C" void launchCube(float * h_in, int n, float * h_out){
    float * d_in;
    float * d_out;
    

    int byteSize = n * sizeof(float);
    
    hipMalloc((void**) &d_in, byteSize);
    hipMalloc((void**) &d_out, byteSize);

    hipMemcpy(d_in, h_in, byteSize, hipMemcpyHostToDevice);

    cube<<<1, n>>>(d_out, d_in);

    hipMemcpy(h_out, d_out, byteSize, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

}




